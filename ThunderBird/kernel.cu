#include "hip/hip_runtime.h"
// Visible Spheres - after Sanders and Kandrot CUDA by Example
// raytrace.cu

#include <chrono>
#include <algorithm>
#include <iostream>
#include <cmath>
#include <cstdlib>
#include <hip/hip_runtime.h>
// to remove intellisense highlighting
#include <>
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include <hip/device_functions.h>

//#define DIM 64
//#define DIMDIM (DIM * DIM)
#define IMG_RES 512
#define NTPB 8
#define M_SPHERES 6
#define RADIUS DIM / 10.0f
#define MIN_RADIUS 2.0f
#define rnd(x) ((float) (x) * rand() / RAND_MAX)
#define INF 2e10f
#define M_PI 3.141592653589793
#define MAX_RAY_DEPTH 5



void checkCUDAError(const char *msg) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		system("pause");
		exit(EXIT_FAILURE);
	}
}

template<typename T>
class Vec3
{
public:
	T x, y, z;
//	__host__ __device__ Vec3() : x(0), y(0), z(0) {}
	__host__ __device__ Vec3(){}
	__host__ __device__ void init(){
		x = 0;
		y = 0;
		z = 0;
	}
	__host__ __device__ void init(T _v){
		x = _v;
		y = _v;
		z = _v;
	}
	__host__ __device__ void init(T _x, T _y, T _z){
		x = _x;
		y = _y;
		z = _z;
	}

	__host__ __device__ Vec3& normalize(){
		T nor2 = length2();
		if (nor2 > 0) {
			T invNor = 1 / sqrt(nor2);
			x *= invNor, y *= invNor, z *= invNor;
		}
		return *this;
	}

	__host__ __device__ Vec3<T> operator * (const T &f) const { 
		Vec3<T> t;
		t.init(x * f, y * f, z * f);
		return t;
	}
	__host__ __device__ Vec3<T> operator * (const Vec3<T> &v) const { 
		Vec3<T> t;
		t.init(x * v.x, y * v.y, z * v.z);
		return t;
	}
	__host__ __device__ T dot(const Vec3<T> &v) const { return x * v.x + y * v.y + z * v.z; }
	__host__ __device__ Vec3<T> operator - (const Vec3<T> &v) const { 
		Vec3<T> t;
		t.init(x - v.x, y - v.y, z - v.z);
		return t;
	}
	__host__ __device__ Vec3<T> operator + (const Vec3<T> &v) const { 
		Vec3<T> t;
		t.init(x + v.x, y + v.y, z + v.z);
		return t;
	}
	__host__ __device__ Vec3<T>& operator += (const Vec3<T> &v) { x += v.x, y += v.y, z += v.z; return *this; }
	__host__ __device__ Vec3<T>& operator *= (const Vec3<T> &v) { x *= v.x, y *= v.y, z *= v.z; return *this; }
//	__host__ __device__ Vec3<T> operator - () const { return Vec3<T>(-x, -y, -z); }
	__host__ __device__ T length2() const { return x * x + y * y + z * z; }
	__host__ __device__ T length() const { return sqrt(length2()); }
	//friend std::ostream & operator << (std::ostream &os, const Vec3<T> &v)
	//{
	//	os << "[" << v.x << " " << v.y << " " << v.z << "]";
	//	return os;
	//}
};

typedef Vec3<float> Vec3f;


class Sphere {
	Vec3f center;                           /// position of the sphere
	float radius, radius2;                  /// sphere radius and radius^2
	Vec3f surfaceColor, emissionColor;      /// surface color and emission (light)
	float transparency, reflection;         /// surface transparency and reflectivity

	//float x, y, z, r;
public:
	Sphere() {}
	void init(Vec3f c,const float r, Vec3f sc, float refl,	float transp, Vec3f ec){
		center = c;
		radius = r;
		radius2 = r*r;
		reflection = refl;
		transparency = transp;
		emissionColor = ec;
		surfaceColor = sc;
	}

	__host__ __device__ Vec3f getCenter() { return center; }
	__host__ __device__ Vec3f getEmissionCr() { return emissionColor; }
	__host__ __device__ Vec3f getSurfaceCr() { return surfaceColor; }
	__host__ __device__ float getTransparency() { return transparency; }
	__host__ __device__ float getReflection() { return reflection; }

	__host__ __device__ bool intersect(const Vec3f &rayorig, const Vec3f &raydir, float &t0, float &t1) const
	{
		Vec3f l = center - rayorig;
		float tca = l.dot(raydir);
		if (tca < 0) return false;
		float d2 = l.dot(l) - tca * tca;

		if (d2 > radius2) return false;
		float thc = sqrt(radius2 - d2);
		t0 = tca - thc;
		t1 = tca + thc;
		return true;
	}

	__host__ __device__ float hit(float ox, float oy) {
		float dx = ox - center.x;
		float dy = oy - center.y;
		if (dx * dx + dy * dy < radius2)
			return sqrtf(radius2 - dx * dx - dy * dy) + center.z;
		else
			return -INF;
	}
};

__constant__ Sphere d_sphere[M_SPHERES];

__host__ __device__ float mix(const float &a, const float &b, const float &mix)
{
	return b * mix + a * (1 - mix);
}

__host__ __device__ void trace(Vec3f rayorig, Vec3f raydir, const int depth, Vec3f* pixel, int k)
{
	float tnear = INFINITY;
	//	const Sphere* sphere = NULL;
	int idx = -1;
	// find intersection of this ray with the sphere in the scene
	for (unsigned i = 0; i < M_SPHERES; ++i) {
		float t0 = INFINITY, t1 = INFINITY;
		if (d_sphere[i].intersect(rayorig, raydir, t0, t1)) {
			if (t0 < 0) t0 = t1;
			if (t0 < tnear) {   // find the closest intersection of speres
				tnear = t0;
				//	sphere = &d_sphere[i];
				idx = i;
			}
		}
	}
	// if there's no intersection return black or background color
	if (idx<0){
		pixel[k].init(1.0f, 0.5f, 0.5f);
		return;
	}
	else{
		Vec3f surfaceColor;
		surfaceColor.init(0);
		Vec3f phit = rayorig + raydir * tnear; // point of intersection
		Vec3f nhit = phit - d_sphere[idx].getCenter(); // normal at the intersection point
		nhit.normalize(); // normalize normal direction

		float bias = 1e-4; // add some bias to the point from which we will be tracing
		bool inside = false;
		if (raydir.dot(nhit) > 0){
			nhit.x = -nhit.x;
			nhit.y = -nhit.y;
			nhit.z = -nhit.z;
			inside = true;
		}
	//	if ((d_sphere[idx].getTransparency() > 0 || d_sphere[idx].getReflection() > 0) && depth < MAX_RAY_DEPTH) {
		//	float facingratio = -raydir.dot(nhit);
			//// change the mix value to tweak the effect
			//float fresneleffect = mix(pow(1 - facingratio, 3), 1, 0.1);
		//	float fresneleffect = 1.0f;
			//// compute reflection direction (not need to normalize because all vectors
			//// are already normalized)
		//	Vec3f refldir = raydir - nhit * 2 * raydir.dot(nhit);
	//		refldir.normalize();
			////Vec3f reflection = trace(phit + nhit * bias, refldir, depth + 1);
			//trace(phit + nhit * bias, refldir, depth + 1, pixel, k);
			//Vec3f refraction;
			//refraction.init(0);
			// if the sphere is also transparent compute refraction ray (transmission)
			//if (d_sphere[idx].getTransparency()) {
			//	float ior = 1.1, eta = (inside) ? ior : 1 / ior; // are we inside or outside the surface?
			//	float cosi = -nhit.dot(raydir);
			//	float k = 1 - eta * eta * (1 - cosi * cosi);
			//	Vec3f refrdir = raydir * eta + nhit * (eta *  cosi - sqrt(k));
			//	refrdir.normalize();
			//	//	refraction = trace(phit - nhit * bias, refrdir, spheres, depth + 1);
			//	trace(phit + nhit * bias, refldir, depth + 1, pixel, k);
			//}
			// the result is a mix of reflection and refraction (if the sphere is transparent)
		//	Vec3f reflection = pixel[k];
		//	surfaceColor = (
		//		reflection * fresneleffect +
		//		refraction * (1 - fresneleffect) * d_sphere[idx].getTransparency()) * d_sphere[idx].getSurfaceCr();

		//	Vec3f refldir = raydir - nhit * 2 * raydir.dot(nhit);
		//	trace(phit + nhit * bias, refldir, depth + 1, pixel, k);
		//	surfaceColor.init(0.0f, 0.0f, 1.0f);
	//	}
	//	else {
			// it's a diffuse object, no need to raytrace any further
			for (unsigned i = 0; i < M_SPHERES; ++i) {
				if (d_sphere[i].getEmissionCr().x > 0) {
					// this is a light
					Vec3f transmission;
					transmission.init(1);
					Vec3f lightDirection = d_sphere[i].getCenter() - phit;
					lightDirection.normalize();
					for (unsigned j = 0; j < M_SPHERES; ++j) {
						if (i != j) {
							float t0, t1;
							if (d_sphere[j].intersect(phit + nhit * bias, lightDirection, t0, t1)) {
								transmission.init(0.7f);
								break;
							}
						}
					}

					float fCoff = nhit.dot(lightDirection);
					if (fCoff < 0)	fCoff = 0.0f;
					surfaceColor += d_sphere[idx].getSurfaceCr() * transmission *	fCoff * d_sphere[i].getEmissionCr();
				}
			//}
		}
		pixel[k] = surfaceColor + d_sphere[idx].getEmissionCr();
		return;
	}
}

__global__ void render(float fov, float viewangle, float aspectratio, float iwidth, float iheight, Vec3f* pixel)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
//	int k = x + y * blockDim.x * gridDim.x;
	int k = x + y * IMG_RES;

	// shared ? //
	float xx = (2 * ((x + 0.5) * iwidth) - 1) * viewangle * aspectratio;
	float yy = (1 - 2 * ((y + 0.5) * iheight)) * viewangle;
	Vec3f raydir, rayorig;
	raydir.init(xx, yy, -1);
	raydir.normalize();
	rayorig.init(0);
	//===========================================//

	// trace //
	trace(rayorig, raydir, 0, pixel, k);
	
	
}



bool SaveImage(char* szPathName, unsigned char* img, int w, int h) {
	// Create a new file for writing
	FILE *f;

	int filesize = 54 + 3 * w*h;  //w is your image width, h is image height, both int

	unsigned char bmpfileheader[14] = { 'B', 'M', 0, 0, 0, 0, 0, 0, 0, 0, 54, 0, 0, 0 };
	unsigned char bmpinfoheader[40] = { 40, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 24, 0 };
	unsigned char bmppad[3] = { 0, 0, 0 };

	bmpfileheader[2] = (unsigned char)(filesize);
	bmpfileheader[3] = (unsigned char)(filesize >> 8);
	bmpfileheader[4] = (unsigned char)(filesize >> 16);
	bmpfileheader[5] = (unsigned char)(filesize >> 24);

	bmpinfoheader[4] = (unsigned char)(w);
	bmpinfoheader[5] = (unsigned char)(w >> 8);
	bmpinfoheader[6] = (unsigned char)(w >> 16);
	bmpinfoheader[7] = (unsigned char)(w >> 24);
	bmpinfoheader[8] = (unsigned char)(h);
	bmpinfoheader[9] = (unsigned char)(h >> 8);
	bmpinfoheader[10] = (unsigned char)(h >> 16);
	bmpinfoheader[11] = (unsigned char)(h >> 24);

	f = fopen("img.bmp", "wb");
	fwrite(bmpfileheader, 1, 14, f);
	fwrite(bmpinfoheader, 1, 40, f);
	for (int i = 0; i<h; i++)
	{
		fwrite(img + (w*(h - i - 1) * 3), 3, w, f);
		fwrite(bmppad, 1, (4 - (w * 3) % 4) % 4, f);
	}
	fclose(f);
	return true;
}

void reportTime(const char* msg, std::chrono::steady_clock::duration span) {
	auto ms = std::chrono::duration_cast<std::chrono::milliseconds>(span);
	std::cout << msg << " - took - " <<
		ms.count() << " millisecs" << std::endl;
}


int main(int argc, char* argv[]) {

	Sphere* h_sphere = new Sphere[M_SPHERES];

	Vec3f center, sc, ec;
	center.init(0.0f, -10004.0f, -20.0f);	sc.init(0.20f, 0.20f, 0.20f);	ec.init(0.0f);
	h_sphere[0].init(center, 10000.0f, sc, 0.0f, 0.0f, ec);

	center.init(0.0, 0, -20);	sc.init(1.00, 0.32, 0.36);	ec.init(0.0f);
	h_sphere[1].init(center, 4.0f, sc, 1.0f, 0.5f, ec);

	center.init(5.0, -1, -15);	sc.init(0.90, 0.76, 0.46);	ec.init(0.0f);
	h_sphere[2].init(center, 2.0f, sc, 1.0f, 0.0f, ec);

	center.init(5.0, 0, -25);	sc.init(0.65, 0.97, 0.97);	ec.init(0.0f);
	h_sphere[3].init(center, 3.0f, sc, 1.0f, 0.0f, ec);

	center.init(-5.5, 0, -15);	sc.init(0.70, 0.90, 0.70);	ec.init(0.0f);
	h_sphere[4].init(center, 3.0f, sc, 1.0f, 0.0f, ec);
			
	// light	
	center.init(10.0, 50, 30);	sc.init(0.00, 0.00, 0.00);	ec.init(2.0f);
	h_sphere[5].init(center, 3.0f, sc, 0.0f, 0.0f, ec);


	hipMemcpyToSymbol(HIP_SYMBOL(d_sphere), h_sphere, sizeof(Sphere) * M_SPHERES);
	delete[] h_sphere;

	// allocate device memory for hit data
	Vec3f* d_a;
	hipMalloc((void**)&d_a, IMG_RES*IMG_RES*sizeof(Vec3f));

	// launch the grid of threads
	dim3 dimGrid(IMG_RES / NTPB, IMG_RES / NTPB);
	dim3 dimBlock(NTPB, NTPB);

	unsigned width = 512, height = 512;
	float invWidth = 1 / float(width), invHeight = 1 / float(height);
	float fov = 45, aspectratio = width / float(height);
	float angle = tan(M_PI * 0.5 * fov / 180.);


	checkCUDAError("pre-raytraceRay error");

	std::chrono::steady_clock::time_point ts, te;
	ts = std::chrono::steady_clock::now();

	render << <dimGrid, dimBlock >> >(fov, angle, aspectratio, invWidth, invHeight, d_a);

	te = std::chrono::steady_clock::now();
	reportTime("Render Time: ", te - ts);

	checkCUDAError("raytraceRay error");


	// copy hit data to host
	Vec3f* h_a = new Vec3f[IMG_RES*IMG_RES];
	hipMemcpy(h_a, d_a, IMG_RES*IMG_RES*sizeof(Vec3f), hipMemcpyDeviceToHost);


	unsigned char* imgbuff = new unsigned char[width*height * 3];
	for (unsigned i = 0; i < width * height; ++i) {
		imgbuff[i * 3] = (unsigned char)(std::min(float(1), h_a[i].x) * 255);
		imgbuff[i * 3 + 1] = (unsigned char)(std::min(float(1), h_a[i].y) * 255);
		imgbuff[i * 3 + 2] = (unsigned char)(std::min(float(1), h_a[i].z) * 255);

	}
	SaveImage("./test.bmp", imgbuff, width, height);

	// clean up
	delete[] imgbuff;
	delete[] h_a;
	hipFree(d_a);
}

